#include "hip/hip_runtime.h"


#include <stdio.h>


// 1 block of threads --> 8 values, grid = 1
__global__ void unique_idx_calc_threadIdx(int * input)
{
    int tid = threadIdx.x;
    printf("threadIdx : %d, value : %d \n", tid, input[tid]);
}

// 4 blocks, each block - 4 threads.
__global__ void unique_gid_calculation(int * input)
{
    int tid = threadIdx.x;
    int offset = blockIdx.x * blockDim.x;
    int gid = tid + offset;

    printf("blockIdx.x : %d, threadIdx.x : %d, blockDim.x : %d, gridDim.x: %d, value : %d \n", blockIdx.x, tid, blockDim.x , gridDim.x, input[gid]);
}

int main()
{
    int array_size = 16;
    int array_bite_size = sizeof(int) * array_size;
    int h_data[] = {23, 9, 4, 53, 64, 12, 1, 33, 22, 11, 9, 12, 13, 89, 90, 77};

    for (int i=0; i < array_size; i++) {
        printf("%d ", h_data[i]);
    }
    printf ("\n \n");

    int * d_data;
    hipMalloc((void **)&d_data, array_bite_size);
    hipMemcpy(d_data, h_data, array_bite_size, hipMemcpyHostToDevice);

    dim3 block(4);
    dim3 grid(4);

    //unique_idx_calc_threadIdx<<<grid, block>>>(d_data);
    unique_gid_calculation<<<grid, block>>>(d_data);
    hipDeviceSynchronize();
    hipDeviceReset();
}
