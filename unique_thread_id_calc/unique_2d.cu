#include "hip/hip_runtime.h"


#include <stdio.h>


// 2 - 2d block of threads --> 4 values in each dimension of x and y , grid = 2
__global__ void unique_gid_calculation2d(int * input)
{
    int tid = blockDim.x * threadIdx.y + threadIdx.x;
    int num_threads_per_block = blockDim.x * blockDim.y;
    int block_offset = blockIdx.x * num_threads_per_block;
    int num_threads_in_row = num_threads_per_block * gridDim.x;
    int row_offset = num_threads_in_row * blockIdx.y;

    int gid = tid + block_offset + row_offset;
    
    printf("blockIdx.x : %d, blockIdx.y : %d, threadIdx.x : %d, gid : %d - data : %d \n", blockIdx.x, blockIdx.y, tid, gid, input[gid]);
}

int main()
{
    int array_size = 16;
    int array_bite_size = sizeof(int) * array_size;
    int h_data[] = {23, 9, 4, 53, 64, 12, 1, 33, 22, 11, 9, 12, 13, 89, 90, 77};

    for (int i=0; i < array_size; i++) {
        printf("%d ", h_data[i]);
    }
    printf ("\n \n");

    int * d_data;
    hipMalloc((void **)&d_data, array_bite_size);
    hipMemcpy(d_data, h_data, array_bite_size, hipMemcpyHostToDevice);

    dim3 block(2,2);
    dim3 grid(2,2);

    //unique_idx_calc_threadIdx<<<grid, block>>>(d_data);
    unique_gid_calculation2d<<<grid, block>>>(d_data);
    hipDeviceSynchronize();
    hipDeviceReset();
}
