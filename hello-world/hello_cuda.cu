#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void hello_cuda()
{
    printf("Hello CUDA world : \n");
}

int main()
{
    //hello_cuda <<<1,10>>>();
    // 
    //dim3 block(4, 1, 1);
    //dim3 grid(8, 1, 1);
    int nx, ny;
    nx = 16;
    ny = 4;

    dim3 block(8, 2);
    dim3 grid(nx/block.x , ny/block.y);

    hello_cuda <<< grid, block >>>();
    hipDeviceSynchronize();

    hipDeviceReset();

    return 0;
}

