#include <stdio.h>
#include <iostream>

#include "hip/hip_runtime.h"

//Kernel code.
__global__ void square(float * d_in, float * d_out)
{
	int idx = threadIdx.x;
	float f = d_in[idx];
	d_out[idx] = f * f;
}


int main()
{
	const int ARRAY_SIZE = 4;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
	
	//input array on host.
	float h_in[ARRAY_SIZE];
	int i;
	for (i=0;i < ARRAY_SIZE; i++) {
		h_in[i] = float(i);
		printf("%f\n", h_in[i]);
	}

	//output array of host.
	float h_out[ARRAY_SIZE];
	
	//Declare GPU memory pointers.
	float * d_in;
	float * d_out;

	//Allocate GPU memory.
	hipMalloc((void **)&d_in, ARRAY_BYTES);
	hipMalloc((void **)&d_out, ARRAY_BYTES);
	
	//transfer the array to GPU.
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	
	square<<<1,ARRAY_SIZE>>>(d_in, d_out);

	//Copy back to host.
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	//Print the results.
	for (int i=0; i < ARRAY_SIZE; i++) {
		printf("%f\n", h_out[i]);
	}

	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
