#include <iostream>
#include <stdio.h>

#include "hip/hip_runtime.h"

__global__ void cubic(const float * d_in, float * d_out)
{
    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f * f * f;
}

int main()
{
    const int ARRAY_SIZE=10;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    float h_in[ARRAY_SIZE];
    for(int i=0;i < ARRAY_SIZE; i++) {
        h_in[i] = float(i);
    }

    float h_out[ARRAY_SIZE];
    
    //Declare GPU memory.
    float * d_in;
    float * d_out;
    
    hipMalloc((void **)&d_in, ARRAY_BYTES);
    hipMalloc((void **)&d_out, ARRAY_BYTES);

    //Tranfer array to GPU.
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    cubic<<<1, ARRAY_SIZE>>>(d_in, d_out);

    //copy bakc the results back to host.
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    //Output the results.
    for(int i=0; i < ARRAY_SIZE; i++) {
        printf("%f\n", h_out[i]);
    }

    hipFree(d_in);
    hipFree(d_out);

    return 0;
}
