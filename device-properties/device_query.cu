#include "hip/hip_runtime.h"


#include <stdio.h>

void query_device()
{
    int device_count = 0;
    hipGetDeviceCount(&device_count);
    
    if (device_count == 0) {
        printf("No CUDA support available \n");
        exit(1);
    }

    int devNo = 0;
    hipDeviceProp_t iProp;
    hipGetDeviceProperties(&iProp, devNo);

    printf(" Device %d: %s \n", devNo, iProp.name);

    hipDeviceReset();
}

int main()
{
    query_device();
    return 0;
}
