#include "hip/hip_runtime.h"



#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <time.h>

/* 
1. three arrays of size 2 ^ 22 randomly initialized.
2. cpu comparision for three arrays sum
3. gpu kernel to sum three arrays. 
4. cuda error mechanism.
5. grid is 1D.
6. check with block size - 64, 128, 256, 512. 

*/

#define CHECK_ERROR(value) { check_cuda_error((value), __FILE__, __LINE__); }
inline void check_cuda_error(hipError_t error, const char * file, int line, bool abort = true)
{
    if (error != hipSuccess) {
        fprintf(stderr, "GPUAssert: %s %s %d \n", hipGetErrorString(error), file, line);
        if (abort) {
            exit(error);
        }
    }
}

// GPU device kernel.
__global__ void sum_three_arrays_gpu(int * a, int * b, int * c, int * d, int size)
{
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id < size) {
        d[global_id] = a[global_id] + b[global_id] + c[global_id];
    }
}

// CPU implementation. 
void sum_three_arrays_cpu(int * a, int * b, int * c, int * d, int size)
{
    for (int i=0; i < size; i++) {
        d[i] = a[i] + b[i] + c[i];
    }
}

bool compare_two_arrays(int * a, int * b, int size)
{
    bool same = true;
    for (int i=0; i < size; i++) {
        if (a[i] != b[i]) {
            same = false;
            break;
        }
    }
    return same;
}

int main(int argc, char * argv[])
{

    int size = 2 << 22;
    int byte_size = size * sizeof(int);
    int block_size = 256;
    std::cout << "Experiment: Block size used is:  " << block_size << std::endl;

    //Create Host arrays.
    int * h_a, *h_b, *h_c, *gpu_results, *cpu_results;
    h_a = (int *)malloc(byte_size);
    h_b = (int *)malloc(byte_size);
    h_c = (int *)malloc(byte_size);
    gpu_results = (int *)malloc(byte_size);
    cpu_results = (int *)malloc(byte_size);

    time_t t;
    srand((unsigned) time(&t));
    
    //Initalize random values for arrays.
    //initialize array a
    for(int i=0; i < size; i++) {
        h_a[i] = (int)(rand() && 0xff);
    }

    //initailize array b
    for(int i=0; i < size; i++) {
        h_a[i] = (int)(rand() && 0xff);
    }

    //initialize array c.
    for(int i=0; i < size; i++) {
        h_a[i] = (int)(rand() && 0xff);
    }

    memset(gpu_results, 0, byte_size);
    memset(cpu_results, 0, byte_size);

    //CPU Results.
    clock_t cpu_start, cpu_end;
    cpu_start = clock();
    sum_three_arrays_cpu(h_a, h_b, h_c, cpu_results, size);
    cpu_end = clock();

    // Allocate device memory.
    int * d_a, *d_b, *d_c, *d_results;
    CHECK_ERROR(hipMalloc((int **)&d_a, byte_size));
    CHECK_ERROR(hipMalloc((int **)&d_b, byte_size));
    CHECK_ERROR(hipMalloc((int **)&d_c, byte_size));
    CHECK_ERROR(hipMalloc((int **)&d_results, byte_size));

    // Move memory host to device.

    clock_t htod_start, htod_end;
    htod_start = clock();
    CHECK_ERROR(hipMemcpy(d_a, h_a, byte_size, hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy(d_b, h_b, byte_size, hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy(d_c, h_c, byte_size, hipMemcpyHostToDevice));
    htod_end = clock();
    
    // Grid and block Size for device execution.
    dim3 block(block_size);
    dim3 grid((size/block.x) + 1);
    
    // Device execution of summation.
    clock_t gpu_start, gpu_end;
    gpu_start = clock();
    sum_three_arrays_gpu<<<grid, block>>>(d_a, d_b, d_c, d_results, size);
    hipDeviceSynchronize();
    gpu_end = clock();
    
    //Device To Host.
    clock_t dtoh_start, dtoh_end;
    dtoh_start = clock();
    CHECK_ERROR(hipMemcpy(gpu_results, d_results, byte_size, hipMemcpyDeviceToHost));
    dtoh_end = clock();
    
    // compare the results.
    auto result = compare_two_arrays(gpu_results, cpu_results, size);
    if (result) {
        printf("Both the CPU and GPU results match. \n");
    } else {
        printf("Mismatch in CPU and GPU results.\n");
    }
    
    // Print execution times.
    printf("Sum array CPU execution time      : %4.6f \n" , (double)((double)(cpu_end - cpu_start)/ CLOCKS_PER_SEC));
    printf("H to D mem transfer time          : %4.6f \n" , (double)((double)(htod_end - htod_start)/ CLOCKS_PER_SEC));
    printf("Sum array GPU execution time      : %4.6f \n" , (double)((double)(gpu_end - gpu_start)/ CLOCKS_PER_SEC));
    printf("D to H mem transfer time          : %4.6f \n" , (double)((double)(dtoh_end - dtoh_start)/ CLOCKS_PER_SEC));
    printf("Sum array GPU total execution time: %4.6f \n" , (double)((double)(dtoh_end - htod_start)/ CLOCKS_PER_SEC));

    CHECK_ERROR(hipFree(d_results));
    CHECK_ERROR(hipFree(d_c));
    CHECK_ERROR(hipFree(d_b));
    CHECK_ERROR(hipFree(d_a));
    
    free(gpu_results);
    free(cpu_results);
    free(h_a);
    free(h_b);
    free(h_c);

    hipDeviceReset();
    return 0;
}
