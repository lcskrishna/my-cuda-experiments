#include "hip/hip_runtime.h"


//#include "cuda_common.h"
#include <stdio.h>
#include <iostream>

#include <stdlib.h>
#include <time.h>

#include <cstring>

__global__ void sum_array(int *a, int * b, int *c, int size)
{
    int tid = threadIdx.x;
    int block_offset = blockDim.x * blockIdx.x;
    int gid = tid + block_offset;

    if (gid < size) {
        c[gid] = a[gid] + b[gid];
    }
}


void compare_arrays(int *a, int *b, int size)
{
    for (int i=0; i < size; i++) {
        if (a[i] != b[i]) {
            std::cout << "Arrays are different" << std::endl;
            return;
        }
    }
    std::cout << "Arrays are same" << std::endl;
}

void sum_array_cpu(int *a, int * b, int * result, int size)
{
    for (int i=0; i < size; i++) {
        result[i] = a[i] + b[i];
    }
}

int main()
{
    int size = 10000;
    int block_size = 128;
    int byte_size = size * sizeof(int);
    
    int * h_a, *h_b;
    int * gpu_results, *h_c;
    
    h_a = (int*)malloc(byte_size);
    h_b = (int*)malloc(byte_size);
    gpu_results = (int*)malloc(byte_size);
    h_c = (int *)malloc(byte_size);
   
    time_t t;
    srand((unsigned)time(&t));
    for (int i=0; i < size; i++) {
        h_a[i] = (int)(rand() & 0xff);
    }
    
    for(int i=0; i < size; i++) {
        h_b[i] = (int) (rand() & 0xff);
    }

    memset(gpu_results, 0, byte_size);

    sum_array_cpu(h_a, h_b, h_c, size);
    
    int * d_a, *d_b;
    int * d_c;
    hipMalloc((void**)&d_a, byte_size);
    hipMalloc((void**)&d_b, byte_size);
    hipMalloc((void**)&d_c, byte_size);

    hipMemcpy(d_a, h_a, byte_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, byte_size, hipMemcpyHostToDevice);

    dim3 block(block_size);
    dim3 grid((size/block.x) + 1);

    sum_array<<<grid, block>>>(d_a, d_b, d_c, size);

    hipDeviceSynchronize();
    hipMemcpy(gpu_results, d_c, byte_size, hipMemcpyDeviceToHost);

    compare_arrays(gpu_results, h_c, size);
   
    hipFree(d_c);
    hipFree(d_b);
    hipFree(d_a);
    
    free(gpu_results);
    free(h_b);
    free(h_a); 
    free(h_c);

    hipDeviceReset();
    return 0;
}
