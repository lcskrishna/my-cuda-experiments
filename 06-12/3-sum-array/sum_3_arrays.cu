#include "hip/hip_runtime.h"


//#include "cuda_common.h"
#include <stdio.h>
#include <iostream>

#include <stdlib.h>
#include <time.h>

#include <cstring>

#define gpuErrCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char * file, int line, bool abort = true)
{
    if (code != hipSuccess) {
        fprintf(stderr, "GPUAssert: %s %s %d \n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void sum_three_array(int *a, int * b, int *c, int *d, int size)
{
    int tid = threadIdx.x;
    int block_offset = blockDim.x * blockIdx.x;
    int gid = tid + block_offset;

    if (gid < size) {
        d[gid] = a[gid] + b[gid] + c[gid];
    }
}


void compare_arrays(int *a, int *b, int size)
{
    for (int i=0; i < size; i++) {
        if (a[i] != b[i]) {
            std::cout << "Arrays are different" << std::endl;
            return;
        }
    }
    std::cout << "Arrays are same" << std::endl;
}

void sum_array_cpu(int *a, int * b, int *c,  int * result, int size)
{
    for (int i=0; i < size; i++) {
        result[i] = a[i] + b[i] + c[i];
    }
}

int main()
{
    int size = 1 << 25;
    int block_size = 128;
    int byte_size = size * sizeof(int);
    
    int * h_a, *h_b, *h_c;
    int * gpu_results, *h_d;
    
    h_a = (int*)malloc(byte_size);
    h_b = (int*)malloc(byte_size);
    gpu_results = (int*)malloc(byte_size);
    h_c = (int *)malloc(byte_size);
    h_d = (int *)malloc(byte_size);
   
    time_t t;
    srand((unsigned)time(&t));
    for (int i=0; i < size; i++) {
        h_a[i] = (int)(rand() & 0xff);
    }
    
    for(int i=0; i < size; i++) {
        h_b[i] = (int) (rand() & 0xff);
    }

    for (int i=0; i < size; i++) {
        h_c[i] = (int)(rand() & 0xff);
    }

    memset(gpu_results, 0, byte_size);

    clock_t cpu_start, cpu_end;
    cpu_start = clock();
    sum_array_cpu(h_a, h_b, h_c, h_d, size);
    cpu_end = clock();
    
    int * d_a, *d_b;
    int * d_c, *d_d;
    gpuErrCheck(hipMalloc((void**)&d_a, byte_size));
    gpuErrCheck(hipMalloc((void**)&d_b, byte_size));
    gpuErrCheck(hipMalloc((void**)&d_c, byte_size));
    gpuErrCheck(hipMalloc((void**)&d_d, byte_size));

    clock_t htod_start, htod_end;
    htod_start = clock();
    gpuErrCheck(hipMemcpy(d_a, h_a, byte_size, hipMemcpyHostToDevice));
    gpuErrCheck(hipMemcpy(d_b, h_b, byte_size, hipMemcpyHostToDevice));
    gpuErrCheck(hipMemcpy(d_c, h_c, byte_size, hipMemcpyHostToDevice));
    htod_end = clock();

    dim3 block(block_size);
    dim3 grid((size/block.x) + 1);

    clock_t gpu_start, gpu_end;
    gpu_start = clock();
    sum_three_array<<<grid, block>>>(d_a, d_b, d_c, d_d, size);
    gpu_end = clock();

    hipDeviceSynchronize();
    clock_t hdtoh_start, hdtoh_end;
    hdtoh_start = clock();
    hipMemcpy(gpu_results, d_d, byte_size, hipMemcpyDeviceToHost);
    hdtoh_end = clock();

    compare_arrays(gpu_results, h_d, size);

    std::cout << "Sum array CPU execution time: " << (double)(((double)(cpu_end - cpu_start)/CLOCKS_PER_SEC)) << std::endl;
    std::cout << "Sum array GPU execution time: " << (double)(((double)(gpu_end - gpu_start)/CLOCKS_PER_SEC)) << std::endl;
    std::cout << "Host to Device Transfer time: " << (double)(((double)(htod_end - htod_start)/CLOCKS_PER_SEC)) << std::endl;
    std::cout << "Device to Host Transfer time: " << (double)(((double)(hdtoh_end - hdtoh_start)/CLOCKS_PER_SEC)) << std::endl;
   
    hipFree(d_d);
    hipFree(d_c);
    hipFree(d_b);
    hipFree(d_a);
    
    free(gpu_results);
    free(h_b);
    free(h_a); 
    free(h_c);
    free(h_d);

    hipDeviceReset();
    return 0;
}
