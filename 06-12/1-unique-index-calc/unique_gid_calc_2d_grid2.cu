#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <iostream>

__global__ void unique_gid_2d(int * input)
{
    int tid = threadIdx.x;
    int block_offset = blockIdx.x * blockDim.x;
    int row_offset = gridDim.x * blockDim.x * blockIdx.y;
    int gid = row_offset + block_offset + tid;

    printf ("blockIdx.x=%d, blockIdx.y=%d, threadIdx.x=%d, gid = %d, value=%d \n", blockIdx.x, blockIdx.y, threadIdx.x, gid, input[gid]);
}

__global__ void unique_gid_2d_2blocks(int * input)
{
    int tid = blockDim.x * threadIdx.y + threadIdx.x;
    int num_threads_block = blockDim.x * blockDim.y;
    int block_offset = blockIdx.x * num_threads_block;
    int num_threads_in_block_row = num_threads_block * gridDim.x ;
    int row_offset = num_threads_in_block_row * blockIdx.y;
    int gid = row_offset + block_offset + tid;

    printf ("blockIdx.x=%d, blockIdx.y=%d, threadIdx.x=%d, threadIdx.y=%d, gid = %d, value=%d \n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, gid, input[gid]);
}

int main()
{
    int array_size = 16;
    int array_byte_size = sizeof(int) * array_size;
    int h_data[] = {23, 9, 4, 53, 65, 12, 1, 33, 87, 45, 23, 12, 342, 56, 44, 99};

    for (int i=0;i < array_size; i++) {
        std::cout << h_data[i] << ", ";
    }
    std::cout << std::endl;

    int *d_data;
    hipMalloc((void **)&d_data, array_byte_size);
    hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

    dim3 block(2,2);
    dim3 grid(2,2);

    unique_gid_2d_2blocks<<<grid, block>>>(d_data);
    hipDeviceSynchronize();
    hipDeviceReset();
    
    return 0;
}
