#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <iostream>

__global__ void unique_idx_calc_threadIdx(int * input)
{
    int tid= threadIdx.x;
    printf ("threadIdx : %d, value: %d \n", tid, input[tid]);
}

__global__ void unique_gid_1d(int * input)
{
    int tid = threadIdx.x;
    int offset = blockIdx.x * blockDim.x;
    int gid = tid + offset;

    printf ("blockIdx.x=%d, threadIdx.x=%d, gid = %d, value=%d \n", blockIdx.x, threadIdx.x, gid, input[gid]);
}

int main()
{
    int array_size = 16;
    int array_byte_size = sizeof(int) * array_size;
    int h_data[] = {23, 9, 4, 53, 65, 12, 1, 33, 87, 45, 23, 12, 342, 56, 44, 99};

    for (int i=0;i < array_size; i++) {
        std::cout << h_data[i] << ", ";
    }
    std::cout << std::endl;

    int *d_data;
    hipMalloc((void **)&d_data, array_byte_size);
    hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

    dim3 block(4);
    dim3 grid(4);

    unique_gid_1d<<<grid, block>>>(d_data);
    hipDeviceSynchronize();
    hipDeviceReset();
    
    return 0;
}
