#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <cstring>
#include <time.h>

__global__ void print_details(int * input)
{
    int tid = threadIdx.x + blockDim.x * threadIdx.y + blockDim.x * blockDim.y * threadIdx.z;
    int num_threads_block = blockDim.x * blockDim.y * blockDim.z;
    int block_offset = num_threads_block * blockIdx.x;
    int num_threads_per_row = num_threads_block * gridDim.x;
    int row_offset = num_threads_per_row * blockIdx.y;
    int num_threads_per_xy = num_threads_block * gridDim.x * gridDim.y;
    int z_offset = num_threads_per_xy * blockIdx.z;
    
    int gid = tid + block_offset + row_offset + z_offset;
    
    printf("tid: %d, gid: %d, value: %d \n", tid, gid, input[gid]);
    
}


int main()
{
    int size = 64;
    int size_in_bytes = size * sizeof(int);
    
    int * h_input;
    h_input = (int*) malloc(size_in_bytes);
    
    // initilize
    time_t t;
    srand((unsigned)time(&t));
    
    for(int i=0; i < size; i++) {
        h_input[i] = (int)(rand() & 0xff);
    }

    for (int i=0; i < size; i++) {
        std::cout << h_input[i] << " ";
    }
    std::cout << std::endl;
    

    // device.
    int * d_input;
    hipMalloc((void**)&d_input, size_in_bytes);
    hipMemcpy(d_input, h_input, size_in_bytes, hipMemcpyHostToDevice);
    
    dim3 block(2,2,2);
    dim3 grid(2,2,2);

    print_details<<<grid, block>>>(d_input);

    hipDeviceSynchronize();
    hipFree(d_input);
    free(h_input);

    hipDeviceReset();
    return 0;

}
