#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void print_details()
{
    printf("threadIdx.x: %d, threadIdx.y: %d, threadIdx.z: %d, blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d, gridDim.x: %d, gridDim.y: %d, gridDim.z: %d \n", threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z);
}

int main()
{
    int nx, ny, nz;
    nx = 4;
    ny = 4;
    nz = 4;
    
    dim3 block(2, 2, 2);
    dim3 grid(nx/block.x, ny/block.y, nz/block.z);

    print_details <<< grid, block>>>();
    hipDeviceReset();

    return 0;
}
