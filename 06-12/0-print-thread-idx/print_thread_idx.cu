#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void print_thread_ids()
{
    printf("threadIdx.x = %d, threadIdx.y: %d, threadIdx.z: %d \n", threadIdx.x, threadIdx.y, threadIdx.z);
} 


int main()
{
    int nx, ny;
    nx = 16;
    ny = 16;


    dim3 block(8, 8);
    dim3 grid(nx/block.x, ny/block.y);

    print_thread_ids <<< grid, block>>>();
    hipDeviceReset();

    return 0;
}
