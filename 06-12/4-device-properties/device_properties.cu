#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>


void query_device()
{
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cout << "ERROR: No CUDA supported device found." << std::endl;
    }
    
    int devNo = 0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, devNo);
    
    printf ("Device %d: %s\n", devNo, prop.name);
    printf ("   Number of Multiprocessors:   %d \n", prop.multiProcessorCount);
    printf ("   Clock rate:   %d \n", prop.clockRate);
    printf ("   Compute Capability:   %d.%d \n", prop.major, prop.minor);
    printf ("   Total amount of global memory:   %4.2f KB\n", prop.totalGlobalMem / 1024.0);
    printf ("   Amount of constant memory:   %4.2f KB\n", prop.totalConstMem / 1024.0);
    
}

int main()
{
    query_device();
    return 0;
}
