#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void print_details_of_wraps()
{
    int gid = (blockIdx.y * gridDim.x * blockDim.x) + (blockDim.x * blockIdx.x) + threadIdx.x;
    int warp_id = threadIdx.x / 32;
    int gbid = blockIdx.y * gridDim.x + blockIdx.x;

    printf("tid: %d, bid.x : %d, bid.y : %d, gid: %d, warp_id : %d, gbid : %d \n", threadIdx.x, blockIdx.x, blockIdx.y, gid, warp_id, gbid);
}

int main(int argc , char ** argv)
{
    dim3 block(42);
    dim3 grid(2, 2);
    
    print_details_of_wraps<<<grid, block>>>();
    hipDeviceSynchronize();
    
    hipDeviceReset();
    return 0;
}
