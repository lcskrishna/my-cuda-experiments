#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <time.h>

__global__ void calculate_unique_3d_idx(int * input, int size)
{
    int tid = (threadIdx.z * blockDim.y * blockDim.x) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int block_id = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int global_index = block_id * blockDim.x * blockDim.y * blockDim.z + tid;

    printf ("tid: %d, block_id : %d, global_index : %d, value: %d \n", tid, block_id, global_index, input[global_index]);
}

int main()
{
    int size = 64;
    int byte_size = sizeof(int) * size;
    
    int * h_data;
    h_data = (int *) malloc(byte_size);
    
    time_t t;
    srand((unsigned) time(&t));
    for (int i =0; i < size; i++) {
        h_data[i] = (int) (rand() && 0xff);
    }

    int * d_data;
    hipMalloc((void **)&d_data, byte_size);
    hipMemcpy(d_data, h_data, byte_size, hipMemcpyHostToDevice);
    

    int nx, ny, nz;
    nx = 4; ny = 4; nz = 4;
    
    dim3 block(2, 2, 2);
    dim3 grid(nx/block.x, ny/block.y, nz/block.z);

    calculate_unique_3d_idx<<<grid, block>>>(d_data, size);
    hipDeviceSynchronize();

    hipFree(d_data);
    free(h_data);
    
    hipDeviceReset();
    return 0;
}
